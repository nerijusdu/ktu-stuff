// IFF-6/11 Nerijus Dulke L4a
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <iostream>
#include <iomanip>
#include <fstream>
#include <string>

using namespace std;

// masyvu skaicius
const int N = 4;
// automobiliu skaicius masyve
const int K = 10;
// maksimalus pavadinimo simboliu skaicius
const int MAX_ILGIS = 15;

struct Automobilis {
public: 
	char pavadinimas[N * MAX_ILGIS + 1];
	int galia;
	double kuroSanaudos;

	// host konstruktorius kvieciamas is CPU
	__host__ Automobilis() : galia(0), kuroSanaudos(0.0) {
		memset(pavadinimas, ' ', N * MAX_ILGIS - 1);
		pavadinimas[N * MAX_ILGIS] = '\0';
	}

	// device konstruktorius kvieciamas is GPU
	__device__ Automobilis(char pavadinimas[], int galia, double kuroSanaudos) {
		for (int i = 0; i < N * MAX_ILGIS; i++)
		{
			this->pavadinimas[i] = pavadinimas[i];
		}
		this->galia = galia;
		this->kuroSanaudos = kuroSanaudos;
	}

	// destruktorius kvieciamas is CPU ir GPU
	__host__ __device__ ~Automobilis() {};
};


// funkcija skirta sudeti masyvu elementu lauku reiksmes
__global__ void sudeti(Automobilis* automobiliai, Automobilis* rezultatai) {
	// paimamas gijos indeksas
	int index = threadIdx.x;
	int galia = 0;
	double kuroSanaudos = 0.0;
	char pavadinimai[N * MAX_ILGIS];

	for (int i = 0; i < N; i++)
	{
		// kadangi duomenys yra viename masyve o ne matricojse,
		// [i * K + index] yra atitinkamas elementas is masyvo
		galia += automobiliai[i * K + index].galia;
		kuroSanaudos += automobiliai[i * K + index].kuroSanaudos;

		for (int j = 0; j < MAX_ILGIS; j++)
		{
			pavadinimai[MAX_ILGIS * i + j] = automobiliai[i * K + index].pavadinimas[j];
		}
	}

	rezultatai[index] = Automobilis(pavadinimai, galia, kuroSanaudos);
}

hipError_t vykdyti(Automobilis** duomenu_matrica, Automobilis* rezultatai) {
	hipError_t status;

	// GPU kintamieji
	Automobilis* device_rezultatai = new Automobilis[K];
	Automobilis* device_duomenys = new Automobilis[K * N];

	// duomenys perkeliami is matricos i masyva
	Automobilis* duomenu_masyvas = new Automobilis[K * N];
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < K; j++)
		{
			duomenu_masyvas[i * K + j] = duomenu_matrica[i][j];
		}
	}

	// Pasirenkamas GPU irenginys
	status = hipSetDevice(0);
	if (status != hipSuccess) {
		fprintf(stderr, "Ivyko klaida pasirenkant GPU");
		goto Error;
	}

	// Paskiriama atmintis GPU
	status = hipMalloc((void**)&device_duomenys, N * K * sizeof(Automobilis));
	if (status != hipSuccess) {
		fprintf(stderr, "Ivyko klaida paskiriant atminti");
		goto Error;
	}
	status = hipMalloc((void**)&device_rezultatai, K * sizeof(Automobilis));
	if (status != hipSuccess) {
		fprintf(stderr, "Ivyko klaida paskiriant atminti");
		goto Error;
	}

	// Nukopijuojami duomenys i GPU
	status = hipMemcpy(device_duomenys, duomenu_masyvas, N * K * sizeof(Automobilis), hipMemcpyHostToDevice);
	if (status != hipSuccess) {
		fprintf(stderr, "Ivyko klaida kopijuojant i GPU");
		goto Error;
	}
	status = hipMemcpy(device_rezultatai, rezultatai, K * sizeof(Automobilis), hipMemcpyHostToDevice);
	if (status != hipSuccess) {
		fprintf(stderr, "Ivyko klaida kopijuojant i GPU");
		goto Error;
	}

	// vykdoma 1 giju bloke, naudojant K giju
	sudeti<<<1, K>>>(device_duomenys, device_rezultatai);

	// patikrinama ar vykdant sudeti atsirado klaidu
	status = hipGetLastError();
	if (status != hipSuccess) {
		fprintf(stderr, "Ivyko klaida vykdant sudeti");
		goto Error;
	}
	
	// laukiama vykdymo pabaigos
	status = hipDeviceSynchronize();
	if (status != hipSuccess) {
		fprintf(stderr, "Ivyko klaida sinchronizuojant");
		goto Error;
	}

	// kuopijuojami rezultatai i CPU
	status = hipMemcpy(rezultatai, device_rezultatai, K * sizeof(Automobilis), hipMemcpyDeviceToHost);
	if (status != hipSuccess) {
		fprintf(stderr, "Ivyko klaida kopijuojant i CPU");
		goto Error;
	}

// ivykus klaidai atlaisvinama atmintis
Error:
	delete[] duomenu_masyvas;
	hipFree(device_duomenys);
	hipFree(device_rezultatai);

	return status;
}

// funkcija skirta skaityti duomenims is failo
void skaityti(Automobilis** automobiliai) {
	ifstream F("IFF_6_11_Dulke_Nerijus_L4.txt");
	string pavadinimas;

	for (int i = 0; i < N; i++)
	{
		Automobilis* automobiliai_temp = new Automobilis[K];
		
		F.ignore();
		for (int j = 0; j < K; j++)
		{
			F >> pavadinimas;
			for (unsigned int k = 0; k < pavadinimas.length(); k++)
			{
				automobiliai_temp[j].pavadinimas[k] = pavadinimas[k];
			}
			F >> automobiliai_temp[j].galia >> automobiliai_temp[j].kuroSanaudos;
			F.ignore();
		}

		automobiliai[i] = automobiliai_temp;
	}

	F.close();
}

// funkcija skirta spausdinti pradinius duomenis i faila
void spausdintiDuomenis(Automobilis** automobiliai) {
	ofstream F("IFF_6_11_Dulke_Nerijus_L4a_rez.txt");
	for (int i = 0; i < N; i++)
	{
		F << "    ----- Automobiliu masyvas Nr. " << (i + 1) << " ----------" << endl;
		F << "   |" << string(MAX_ILGIS, '-') << "|" << string(12, '-') << "|" << string(13, '-') << "|" << endl;
		F << "   |" << setw(MAX_ILGIS) << left << "Pavadinimas" << setw(13) << left << "|Galia" << setw(9) << left << "|Kuro sanaudos|" << endl;
		F << "   |" << string(MAX_ILGIS, '-') << "|" << string(12, '-') << "|" << string(13, '-') << "|" << endl;
		for (int j = 0; j < K; j++) {
			F << setw(3) << left << (j + 1) << "|";
			for (int k = 0; k < MAX_ILGIS; k++) F << automobiliai[i][j].pavadinimas[k];
			F << "|" << setw(12) << left << automobiliai[i][j].galia << "|";
			F << setw(13) << left << fixed << setprecision(2) << automobiliai[i][j].kuroSanaudos << "|" << endl;
		}
		F << "   |" << string(MAX_ILGIS, '-') << "|" << string(12, '-') << "|" << string(13, '-') << "|" << endl;
		F << endl;
	}
}

// funkcija skirta spausdinti rezultatus i faila
void spausdintiRezultatus(Automobilis* automobiliai) {
	ofstream F("IFF_6_11_Dulke_Nerijus_L4a_rez.txt", ios::app);
	int lineNr = 1;
	F << "   ************" << endl;
	F << "    Rezultatai" << endl;
	F << "   ************" << endl;
	F << "   |" << string((N * MAX_ILGIS), '-') << "|" << string(12, '-') << "|" << string(13, '-') << "|" << endl;
	F << "   |" << setw(N * MAX_ILGIS) << left << "Sujungti pavadinimai" << setw(13) << left << "|Galia" << setw(9) << left << "|Kuro sanaudos|" << endl;
	F << "   |" << string((N * MAX_ILGIS), '-') << "|" << string(12, '-') << "|" << string(13, '-') << "|" << endl;
	for (int i = 0; i < K; i++) {
		F << setw(3) << left << lineNr++ << "|" << setw(N * MAX_ILGIS) << left << automobiliai[i].pavadinimas;
		F << "|" << setw(12) << left << automobiliai[i].galia << "|";
		F << setw(13) << left << fixed << setprecision(2) << automobiliai[i].kuroSanaudos << "|" << endl;
	}
	F << "   |" << string((N * MAX_ILGIS), '-') << "|" << string(12, '-') << "|" << string(13, '-') << "|" << endl;
	F.close();
}

int main() {
	// dvimatis duomenu masyvas
	Automobilis** automobiliai = new Automobilis*[N];
	Automobilis* rezultatai = new Automobilis[K];

	skaityti(automobiliai);

	// vykdom pagrindine funkcija ir tikrinama ar neivyko klaidu
	hipError_t status = vykdyti(automobiliai, rezultatai);
	if (status != hipSuccess) {
		fprintf(stderr, "Ivyko klaida");
		return 1;
	}

	//atspausdinami duomenys ir rezultatai
	spausdintiDuomenis(automobiliai);
	spausdintiRezultatus(rezultatai);

	// atlaisvinama atmintis
	delete[] automobiliai;
	delete[] rezultatai;

	return 0;
}
