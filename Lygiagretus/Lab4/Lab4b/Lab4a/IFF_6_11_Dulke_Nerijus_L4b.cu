// IFF-6/11 Nerijus Dulke L4b
#include <stdio.h>
#include <iostream>
#include <iomanip>
#include <fstream>
#include <string>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

using namespace std;

// masyvu skaicius
const int N = 4;
// automobiliu skaicius masyve
const int K = 10;
// maksimalus pavadinimo simboliu skaicius
const int MAX_ILGIS = 15;

struct Automobilis {
public: 
	char pavadinimas[N * MAX_ILGIS + 1];
	int galia;
	double kuroSanaudos;

	// konstruktorius kvieciamas is CPU arba GPU
	__host__ __device__ Automobilis() : galia(0), kuroSanaudos(0.0) {
		memset(pavadinimas, ' ', N * MAX_ILGIS - 1);
		pavadinimas[N * MAX_ILGIS] = '\0';
	};

	// konstruktorius kvieciamas is CPU arba GPU
	__device__ __host__ Automobilis(char pavadinimas[], int galia, double kuroSanaudos) {
		for (int i = 0; i < N * MAX_ILGIS; i++)
		{
			this->pavadinimas[i] = pavadinimas[i];
		}
		this->galia = galia;
		this->kuroSanaudos = kuroSanaudos;
	};

	// destruktorius kvieciamas is CPU arba GPU
	__host__ __device__ ~Automobilis() {};
};


// funkcija skirta sudeti masyvu elementu lauku reiksmes
Automobilis sudeti(int id, thrust::device_vector<Automobilis>::iterator dev_iter_start) {
	// pradzios iteratoriui priskiriama atitinkamas vektoriaus elementas
	thrust::device_vector<Automobilis>::iterator iter = dev_iter_start + id;
	int galia = 0;
	double kuroSanaudos = 0.0;
	char pavadinimai[N * MAX_ILGIS];

	for (int i = 0; i < N; i++)
	{
		// paimamas automobilio objektas
		Automobilis temp = (static_cast<Automobilis>(*iter));

		galia += temp.galia;
		kuroSanaudos += temp.kuroSanaudos;
		for (int j = 0; j < MAX_ILGIS; j++)
		{
			pavadinimai[MAX_ILGIS * i + j] = temp.pavadinimas[j];
		}

		// iteratorius pereina i kita eilute (kuri yra uz K poziciju)
		iter += K;
	}

	return Automobilis(pavadinimai, galia, kuroSanaudos);
}

// funkcija skirta skaityti duomenims is failo
void skaityti(thrust::host_vector<Automobilis> &automobiliai) {
	ifstream F("IFF_6_11_Dulke_Nerijus_L4.txt");
	string pavadinimas;

	for (int i = 0; i < N; i++)
	{
		F.ignore();
		for (int j = 0; j < K; j++)
		{
			Automobilis automobilis_temp = Automobilis();

			F >> pavadinimas;
			for (unsigned int k = 0; k < pavadinimas.length(); k++)
			{
				automobilis_temp.pavadinimas[k] = pavadinimas[k];
			}
			F >> automobilis_temp.galia >> automobilis_temp.kuroSanaudos;
			automobiliai.push_back(automobilis_temp);
			
			F.ignore();
		}
	}

	F.close();
}

// funkcija skirta spausdinti pradinius duomenis i faila
void spausdintiDuomenis(thrust::host_vector<Automobilis> &automobiliai) {
	ofstream F("IFF_6_11_Dulke_Nerijus_L4b_rez.txt");
	for (int i = 0; i < N; i++)
	{
		F << "   ------ Automobiliu masyvas Nr. " << (i + 1) << " ----------" << endl;
		F << "   |" << string(MAX_ILGIS, '-') << "|" << string(12, '-') << "|" << string(13, '-') << "|" << endl;
		F << "   |" << setw(MAX_ILGIS) << left << "Pavadinimas" << setw(13) << left << "|Galia" << setw(9) << left << "|Kuro sanaudos|" << endl;
		F << "   |" << string(MAX_ILGIS, '-') << "|" << string(12, '-') << "|" << string(13, '-') << "|" << endl;
		for (int j = 0; j < K; j++) {
			F << setw(3) << left << (j + 1) << "|";
			for (int k = 0; k < MAX_ILGIS; k++) F << automobiliai[i * K + j].pavadinimas[k];
			F << "|" << setw(12) << left << automobiliai[i * K + j].galia << "|";
			F << setw(13) << left << fixed << setprecision(2) << automobiliai[i * K + j].kuroSanaudos << "|" << endl;
		}
		F << "   |" << string(MAX_ILGIS, '-') << "|" << string(12, '-') << "|" << string(13, '-') << "|" << endl;
		F << endl;
	}
}

// funkcija skirta spausdinti rezultatus i faila
void spausdintiRezultatus(thrust::host_vector<Automobilis> &automobiliai) {
	ofstream F("IFF_6_11_Dulke_Nerijus_L4b_rez.txt", ios::app);
	F << "   ************" << endl;
	F << "    Rezultatai" << endl;
	F << "   ************" << endl;
	F << "   |" << string((N * MAX_ILGIS), '-') << "|" << string(12, '-') << "|" << string(13, '-') << "|" << endl;
	F << "   |" << setw(N * MAX_ILGIS) << left << "Sujungti pavadinimai" << setw(13) << left << "|Galia" << setw(9) << left << "|Kuro sanaudos|" << endl;
	F << "   |" << string((N * MAX_ILGIS), '-') << "|" << string(12, '-') << "|" << string(13, '-') << "|" << endl;
	for (int i = 0; i < K; i++) {
		F << setw(3) << left << (i + 1) << "|";// << setw(N * MAX_ILGIS) << left << automobiliai[i].pavadinimas;
		for (int j = 0; j < N * MAX_ILGIS; j++) {
			F << automobiliai[i].pavadinimas[j];
		}
		F << "|" << setw(12) << left << automobiliai[i].galia << "|";
		F << setw(13) << left << fixed << setprecision(2) << automobiliai[i].kuroSanaudos << "|" << endl;
	}
	F << "   |" << string((N * MAX_ILGIS), '-') << "|" << string(12, '-') << "|" << string(13, '-') << "|" << endl;
	F.close();
}

int main() {
	// sukuriami CPU (host) ir GPU (device) duomenu ir rezultatu vektoriai
	thrust::host_vector<Automobilis> automobiliai;
	thrust::host_vector<Automobilis> rezultatai;
	thrust::device_vector<Automobilis> dev_automobiliai;
	thrust::device_vector<Automobilis> dev_rezultatai(K);

	// nuskaitomi pradiniai duomenys
	skaityti(automobiliai);

	// duomenys nukopijuojami is CPU i GPU atminti
	dev_automobiliai = automobiliai;

	// sukuriamas GPU atmintyje esanciu duomenu pradzios iteratorius
	thrust::device_vector<Automobilis>::iterator dev_iter_start = dev_automobiliai.begin();

	// sujungiami kiekvieno proceso duomenu laukai 
	for (int i = 0; i < K; i++)
	{
		dev_rezultatai[i] = sudeti(i, dev_iter_start);
	}

	// rezultatai kopijuojami atgal is GPU i CPU atminti
	rezultatai = dev_rezultatai;

	// atspausdinami pradiniai duomenys ir rezultatai
	spausdintiDuomenis(automobiliai);
	spausdintiRezultatus(rezultatai);

	return 0;
}
